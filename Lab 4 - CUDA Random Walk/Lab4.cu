#include "hip/hip_runtime.h"
/*
Author: Yashwanth Soogoor
Class: ECE6122 Section A
Last Date Modified: 11/7/2023
Description:
This file implements a CUDA random walk by talking in the number of walkers and number of steps using three different memory models: pageable, pinned, and managed.
For each model, it calculates the average distance of the walkers from the starting position and the time to calculate in microseconds. 
*/




#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <cstdint>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Function to interrogate command line arguments
bool interrogateCommandLine(int commandLineLength, char* commandLineArguments[], uint64_t& walkers, uint64_t& steps);

// Function to validate numerical input
bool validInput(const std::string& input, uint64_t& value);

// CUDA kernel to calculate random walk using hiprand for random direction generation
// Updates the positions of the walkers and takes in the number of walkers and number of steps and seed for random direction generation
__global__ void randomWalk(uint64_t* position, uint64_t walkers, uint64_t steps, unsigned long long seed) 
{
    int walkerId = blockIdx.x * blockDim.x + threadIdx.x;
    if (walkerId < walkers) 
    {
        int x = 0, y = 0;

        // Start hiprand for random direction generation
        hiprandState state;
        hiprand_init(seed, walkerId, 0, &state);

        for (uint64_t step = 0; step < steps; step++) 
        {
            // Generate a random float between 0 and 1
            float randomValue = hiprand_uniform(&state);

            // Determine the direction that the walker goes next according to the random float
            if (randomValue < 0.25f) x--; // Left
            else if (randomValue < 0.5f) x++; // Right
            else if (randomValue < 0.75f) y++; // Up
            else y--; // Down
        }

        //update x and y values of position
        position[walkerId * 2] = x;
        position[walkerId * 2 + 1] = y;
    }
}


//The main function returns 0 on finishing and exits on 1 if there is error or invalid entries by taking in the command line length and arguments
int main(int commandLineLength, char* commandLineArguments[]) 
{

    uint64_t walkers;
    uint64_t steps;   

    // Interrogate command-line for valid arguments and update the number of walkers/steps if entered, otherwise use defaults 
    if (!interrogateCommandLine(commandLineLength, commandLineArguments, walkers, steps)) 
    {
        return 1; // Exit if not valid command line entries 
    }



    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    //Pageable Warm-Up; Used to warm-up 

    // Memory allocation on host

    uint64_t* positionHostPageablewarmup;
    positionHostPageablewarmup = (uint64_t*)malloc(2 * walkers * sizeof(uint64_t));

    // Initialize host position array
    memset(positionHostPageablewarmup, 0, 2 * walkers * sizeof(uint64_t));

    // Memory allocation on device
    uint64_t* positionPageablewarmup = nullptr;
    hipMalloc((void**)&positionPageablewarmup, 2 * walkers * sizeof(uint64_t));


    // Tell kernel to do random walk

    int numThreadswarmup = 256;
    int numBlockswarmup = (walkers + numThreadswarmup - 1) / numThreadswarmup;
    randomWalk << <numBlockswarmup, numThreadswarmup >> > (positionPageablewarmup, walkers, steps, time(NULL));

    // Transfer position to the host
    hipMemcpy(positionHostPageablewarmup, positionPageablewarmup, 2 * walkers * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // Calculate average distance for Pageable CUDA memory allocation
    double totalDistancePageablewarmup = 0.0;
    for (int i = 0; i < walkers; i++) 
    {
        int x = positionHostPageablewarmup[i * 2];
        int y = positionHostPageablewarmup[i * 2 + 1];
        totalDistancePageablewarmup += sqrt(x * x + y * y);
    }
    double averageDistancePageablewarmup = totalDistancePageablewarmup / walkers;

    // Free memory for Pageable CUDA memory allocation
    hipFree(positionPageablewarmup);
    free(positionHostPageablewarmup);






    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    //Pageable

    // Start measuring time for Pageable CUDA memory allocation
    auto startPageable = std::chrono::high_resolution_clock::now();

    // Memory allocation on host

    uint64_t* positionHostPageable;
    positionHostPageable = (uint64_t*)malloc(2 * walkers * sizeof(uint64_t));

    // Initialize host position array
    memset(positionHostPageable, 0, 2 * walkers * sizeof(uint64_t));

    // Memory allocation on device
    uint64_t* positionPageable = nullptr;
    hipMalloc((void**)&positionPageable, 2 * walkers * sizeof(uint64_t));


    // Kernel for random walk 

    int numThreads = 256;
    int numBlocks = (walkers + numThreads - 1) / numThreads;
    randomWalk << <numBlocks, numThreads >> > (positionPageable, walkers, steps, time(NULL));

    // Transfer position to host
    hipMemcpy(positionHostPageable, positionPageable, 2 * walkers * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // Calculate average distance for Pageable CUDA memory allocation
    double totalDistancePageable = 0.0;
    for (int i = 0; i < walkers; i++) 
    {
        int x = positionHostPageable[i * 2];
        int y = positionHostPageable[i * 2 + 1];
        totalDistancePageable += sqrt(x * x + y * y);
    }
    double averageDistancePageable = totalDistancePageable / walkers;

    // Free memory for Pageable CUDA memory allocation
    hipFree(positionPageable);
    free(positionHostPageable);

    // End time for Pageable CUDA memory allocation
    auto endPageable = std::chrono::high_resolution_clock::now();


    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    //Pinned

    // Start time for pinned CUDA memory allocation
    auto startPinned = std::chrono::high_resolution_clock::now();

    // Memory allocation on host using hipHostMalloc
    uint64_t* positionHostPinned = nullptr;
    hipHostMalloc((void**)&positionHostPinned, 2 * walkers * sizeof(uint64_t));

    // Initialize host position array
    memset(positionHostPinned, 0, 2 * walkers * sizeof(uint64_t));

    // Memory allocation on device
    uint64_t* positionPinned = nullptr;
    hipMalloc((void**)&positionPinned, 2 * walkers * sizeof(uint64_t));

    // Transfer position from host to device
    hipMemcpy(positionPinned, positionHostPinned, 2 * walkers * sizeof(uint64_t), hipMemcpyHostToDevice);

    // Kernel for random walk for pinned CUDA memory allocation
    randomWalk << <numBlocks, numThreads >> > (positionPinned, walkers, steps, time(NULL));

    // Transfer position from device to host
    hipMemcpy(positionHostPinned, positionPinned, 2 * walkers * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // Calculate average distance for pinned CUDA memory allocation
    double totalDistancePinned = 0.0;
    for (int i = 0; i < walkers; i++) 
    {
        int x = positionHostPinned[i * 2];
        int y = positionHostPinned[i * 2 + 1];
        totalDistancePinned += sqrt(x * x + y * y);
    }
    double averageDistancePinned = totalDistancePinned / walkers;

    // Free memory for pinned CUDA memory allocation
    hipFree(positionPinned);
    hipHostFree(positionHostPinned);

    // End time for pinned CUDA memory allocation
    auto endPinned = std::chrono::high_resolution_clock::now();


    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    //Managed

    // Start time for managed CUDA memory allocation
    auto startManaged = std::chrono::high_resolution_clock::now();

    // Memory allocation using hipMallocManaged
    uint64_t* positionManaged = nullptr;
    hipMallocManaged((void**)&positionManaged, 2 * walkers * sizeof(uint64_t));

    // Initialize host position array
    memset(positionManaged, 0, 2 * walkers * sizeof(uint64_t));

    // Launch kernel for managed CUDA memory allocation
    randomWalk << <numBlocks, numThreads >> > (positionManaged, walkers, steps, time(NULL));

    // Synchronize
    hipDeviceSynchronize();

    // Calculate average distance for managed CUDA memory allocation
    double totalDistanceManaged = 0.0;
    for (int i = 0; i < walkers; i++) 
    {
        int x = positionManaged[i * 2];
        int y = positionManaged[i * 2 + 1];
        totalDistanceManaged += sqrt(x * x + y * y);
    }
    double averageDistanceManaged = totalDistanceManaged / walkers;

    // Free memory for managed CUDA memory allocation
    hipFree(positionManaged);

    // End time for managed CUDA memory allocation
    auto endManaged = std::chrono::high_resolution_clock::now();

    // Calculate the time for Pageable CUDA memory allocation
    std::chrono::duration<double, std::micro> totalTimePageable = endPageable - startPageable;

    // Calculate the time for pinned CUDA memory allocation
    std::chrono::duration<double, std::micro> totalTimePinned = endPinned - startPinned;

    // Calculate the time for managed CUDA memory allocation
    std::chrono::duration<double, std::micro> totalTimeManaged = endManaged - startManaged;

    // Output the results
    std::cout << "Lab4 -W " << walkers << " -I " << steps << std::endl;
    std::cout << " Normal CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << totalTimePageable.count() << std::endl;
    std::cout << "    Average distance from origin: " << averageDistancePageable << std::endl;
    std::cout << " Pinned CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << totalTimePinned.count() << std::endl;
    std::cout << "    Average distance from origin: " << averageDistancePinned << std::endl;
    std::cout << " Managed CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << totalTimeManaged.count() << std::endl;
    std::cout << "    Average distance from origin: " << averageDistanceManaged << std::endl;
    std::cout << "Bye" << std::endl;

    return 0;
}

//This function interrogrates command line entries; Inputs are the number of entries and character pointers of the entries and the number of walkers and steps are passed in by reference
//True is returned for valid entries and false is returned if there are invalid entries
bool interrogateCommandLine(int commandLineLength, char* commandLineArguments[], uint64_t& walkers, uint64_t& steps) 
{
    bool walkersEntered = false;
    bool stepsEntered = false;

    for (int i = 1; i < commandLineLength; i++) 
    {
        std::string arg = commandLineArguments[i];
        if (arg == "-W" && i + 1 < commandLineLength) 
        {
            if (validInput(commandLineArguments[i + 1], walkers)) 
            {
                walkersEntered = true;
                continue;
            }
            else 
            {
                std::cerr << "Invalid input for the number of walkers." << std::endl;
                return false;
            }
        }
        else if (arg == "-I" && i + 1 < commandLineLength) 
        {
            if (validInput(commandLineArguments[i + 1], steps)) 
            {
                stepsEntered = true;
                continue;
            }
            else 
            {
                std::cerr << "Invalid input for the number of steps." << std::endl;
                return false;
            }
        }
    }

    // Set defaults if not set in command line
    if (!walkersEntered) 
    {
        walkers = 1000; 
    }
    if (!stepsEntered) 
    {
        steps = 10000; 
    }

    return true;
}

//This checks if the inputs for the number of walkers and steps are valid numbers and takes them by reference and outputs true for valid input and false for invalid input
bool validInput(const std::string& input, uint64_t& value) 
{
    size_t loc; // To check where conversion stops

    // Try to convert the input string to an unsigned 64-bit integer
    value = std::stoull(input, &loc, 10);

    // Check if string was successfully converted
    if (loc == input.length() && value > 0) 
    {
        return true; // Input is valid
    }

    return false; // Input is invalid
}
